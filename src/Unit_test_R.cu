#include "Unit_test.cuh"
#include <iostream>
#include <ctime>
using namespace std;



void test_matrixUpload() {
#include "read_test_data"
	double offset[] = { 10 };
	double* address = new double[9];
	upload(data, rowInd, colInd, size, offset, address);
	double* D_data = new double[(LARGEINDEX)size[0]];
	double* D_rowInd = new double[(LARGEINDEX)size[1]];
	double* D_colInd = new double[(LARGEINDEX)size[2]];
	download(D_data, D_rowInd, D_colInd, address);
	double error = 0;
	error+=checkValue(data, D_data,size[0]);
	error += checkValue(D_rowInd, D_rowInd, size[1]);
	error += checkValue(D_colInd, D_colInd, size[2]);

	if (error == 0) {
		std::cout << "Matrix upload and download test report: Pass" << std::endl;
	}
	else {
		std::cout << "Matrix upload and download test report: Error is : " << error << std::endl;
	}
	double* result = new double[10];
	int a = 1;
	colSums(&a, address, result);
	//print_partial_matrix("C cuda kernel:", result, 1, 10);
	error = checkValue(result, rowsum, size[0]);
	if (error == 0) {
		std::cout << "Matrix rowsum test report: Pass" << std::endl;
	}
	else {
		std::cout << "Matrix rowsum test report: Error is : " << error << std::endl;
	}
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;


}
