#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include "CommonHeader.cuh"
#include <hipsparse.h>
#include "Test_tools.cuh"
#include "MatrixOperation.cuh"
#include "Unit_test.cuh"

/*
* Generate random dense matrix A in column-major order, while rounding some
* elements down to zero to ensure it is sparse.
*/
template<class T>
int generate_random_dense_matrix(int M, int N, T **outA)
{
	int i, j;
	double rMax = (double)RAND_MAX;
	T *A = (T *)malloc(sizeof(T) * M * N);
	int totalNnz = 0;
	double progress = 0;
	for (j = 0; j < N; j++)
	{
		for (i = 0; i < M; i++)
		{
			int r = rand();
			T *curr = A + (j * M + i);

			if (r % 100 >= 50)
			{
				*curr = 0.0f;
			}
			else
			{
				double dr = (double)r;
				*curr = (int)(dr / rMax * 100.0);
			}

			if (*curr != 0.0f)
			{
				totalNnz++;
			}
		}
		if (j > N*progress) {
			//std::cout << "Progress : " << progress << std::endl;
			progress = progress + 0.05;
		}
	}

	*outA = A;
	return totalNnz;
}


template<class T>
T* CPU_matrixSum(hipsparseHandle_t handle, int M, int N, int totalANnz, int *dANnzPerCol, T *dCscValA, int *dCscRowPtrA, int *dCscColIndA, bool isCol = false) {
	int K1;
	int K2;
	hipsparseOperation_t op;
	if (isCol) {
		//Col sum
		K1 = M;
		K2 = N;
	}
	else {
		//Row sum
		K1 = N;
		K2 = M;
	}
	float *C = new float[K2];

	//Download the CSC data
	float *hCscValA = new float[totalANnz];
	int *hCscRowPtrA = new int[totalANnz];
	int *hCscColIndA = new int[N + 1];
	hipMemcpy(hCscValA, dCscValA, sizeof(*dCscValA) * totalANnz, hipMemcpyDeviceToHost);
	hipMemcpy(hCscRowPtrA, dCscRowPtrA, sizeof(*dCscRowPtrA) * totalANnz, hipMemcpyDeviceToHost);
	hipMemcpy(hCscColIndA, dCscColIndA, sizeof(*dCscColIndA) * (N + 1), hipMemcpyDeviceToHost);


	/*
	print_partial_matrix("D:", hCscValA, 1, totalANnz);
	print_partial_matrix("D:", hCscRowPtrA, 1, totalANnz);
	print_partial_matrix("D:", hCscColIndA, 1, N + 1);*/

	clock_t begin = clock();
	for (int i = 0; i < K2; i++) {
		C[i] = 0;
	}
	if (isCol) {
		for (int colID = 0; colID < N; colID++) {
			int nonzero = hCscColIndA[colID + 1] - hCscColIndA[colID];
			for (int i = 0; i < nonzero; i++) {
				C[colID] = C[colID] + hCscValA[hCscColIndA[colID] + i];
			}
		}
	}
	else {
		int rowID;
		for (int colID = 0; colID < N; colID++) {
			int nonzero = hCscColIndA[colID + 1] - hCscColIndA[colID];
			for (int i = 0; i < nonzero; i++) {
				rowID = hCscRowPtrA[hCscColIndA[colID] + i];
				C[rowID] = C[rowID] + hCscValA[hCscColIndA[colID] + i];
			}
		}
	}
	clock_t end = clock();
	double timeSec = (end - begin) / static_cast<double>(CLOCKS_PER_SEC);
	std::cout << "CPU Elapsed time: " << timeSec << std::endl;
	return C;
}

void test_colsum(int M,int N,bool isCol) {
	int K1;
	int K2;
	hipsparseOperation_t op;
	if (isCol) {
		//Col sum
		op = hipsparseOperation_t::HIPSPARSE_OPERATION_NON_TRANSPOSE;
		K1 = M;
		K2 = N;
	}
	else {
		//Row sum
		op = hipsparseOperation_t::HIPSPARSE_OPERATION_TRANSPOSE;
		K1 = N;
		K2 = M;
	}

	// Create the cuSPARSE handle
	hipsparseHandle_t handle = 0;
	hipsparseCreate(&handle);

	// Generate input
	srand(9384);
	float *A, *dA;
	int trueANnz = generate_random_dense_matrix(M, N, &A);
	//print_partial_matrix("A:",A, M, N, M, N);

	// Allocate device memory for vectors and the dense form of the matrix A
	hipMalloc((void **)&dA, sizeof(*A) * M * N);

	// Construct a descriptor of the matrix A
	hipsparseMatDescr_t Adescr = 0;
	hipsparseCreateMatDescr(&Adescr);
	hipsparseSetMatType(Adescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(Adescr, HIPSPARSE_INDEX_BASE_ZERO);
	// Transfer the input vectors and dense matrix A to the device
	hipMemcpy(dA, A, sizeof(*A) * M * N, hipMemcpyHostToDevice);

	// Compute the number of non-zero elements in A
	int totalANnz;
	int *dANnzPerCol;
	hipMalloc((void **)&dANnzPerCol, sizeof(int) * N);
	hipsparseSnnz(handle, HIPSPARSE_DIRECTION_COLUMN, M, N, Adescr,
		dA, M, dANnzPerCol, &totalANnz);

	if (totalANnz != trueANnz)
	{
		fprintf(stderr, "Difference detected between cuSPARSE NNZ and true "
			"value: expected %d but got %d\n", trueANnz, totalANnz);
		return;
	}

	// Allocate device memory to store the sparse CSC representation of A
	float *dCscValA;
	int *dCscRowPtrA;
	int *dCscColIndA;
	hipMalloc((void **)&dCscValA, sizeof(*A) * totalANnz);
	hipMalloc((void **)&dCscRowPtrA, sizeof(*dCscRowPtrA) * totalANnz);
	hipMalloc((void **)&dCscColIndA, sizeof(*dCscColIndA) * (N + 1));

	// Convert A from a dense formatting to a CSR formatting, using the GPU
	hipsparseSdense2csc(handle, M, N, Adescr, dA, M, dANnzPerCol,
		dCscValA, dCscRowPtrA, dCscColIndA);
	//Delete the unused matrix
	free(A);
	hipFree(dA);
	//std::cout << hipGetErrorString(hipGetLastError()) << std::endl;


	float *C_cpu = CPU_matrixSum(handle, M, N, totalANnz, dANnzPerCol, dCscValA, dCscRowPtrA, dCscColIndA, isCol);
	//print_partial_matrix("C cpu:", C_cpu, 1, K2);


	float * C_cuda = new float[K2];
	//CUDA_matrixSum(M, N, totalANnz, dCscValA, dCscRowPtrA, dCscColIndA, C_cuda,isCol);
	//print_partial_matrix("C cuda:", C_cuda, 1, K2);


	float *C_cuda_kernel = new float[K2];
	fillWithNum(C_cuda_kernel, (float)0, K2);
	float *dev_C_cuda = 0;
	hipMalloc((void **)&dev_C_cuda, sizeof(*dev_C_cuda) * K2);
	hipMemcpy(dev_C_cuda, C_cuda_kernel, sizeof(*dev_C_cuda) * K2, hipMemcpyHostToDevice);

	clock_t begin = clock();
	CUDA_matrixSum_kernel << <BlockNum, ThreadNum >> >(M, N, totalANnz, dCscValA, dCscRowPtrA, dCscColIndA, dev_C_cuda, isCol);
	clock_t end = clock();
	double timeSec = (end - begin) / static_cast<double>(CLOCKS_PER_SEC);
	std::cout << "GPU Elapsed time: " << timeSec << std::endl;
	hipDeviceSynchronize();
	hipMemcpy(C_cuda_kernel, dev_C_cuda, sizeof(*dev_C_cuda) * K2, hipMemcpyDeviceToHost);
	//print_partial_matrix("C cuda kernel:", C_cuda_kernel, 1, K2);
	hipFree(dev_C_cuda);

	double error;
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	
	error = checkValue(C_cpu, C_cuda_kernel, K2);
	if (error <= 0.001&&error>=-0.001) {
		std::cout << "Matrix colsum test report: Pass" << std::endl;
	}
	else {
		std::cout << "Matrix colsum test report: Error is : " << error << std::endl;
	}

	delete[] C_cuda;
	delete[] C_cuda_kernel;
	delete[] C_cpu;

	hipsparseDestroyMatDescr(Adescr);
	hipFree(dANnzPerCol);
	hipFree(dCscValA);
	hipFree(dCscRowPtrA);
	hipFree(dCscColIndA);
	hipsparseDestroy(handle);
}