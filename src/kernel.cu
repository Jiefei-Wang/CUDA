
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <ctime>
#include <hipsparse.h>


#include "SparseMatrix.cuh"
#include "Unit_test.cuh"
#include "MatrixOperation.cuh"

#include "Rfuncs.cuh"
#include "Test_tools.cuh"






int main(int argc, char **argv)
{
	bool isCol = false;

	int M = 10000;
	int N = 10000;
	test_colsum(M,N,isCol);
	//test_matrixUpload();
	
	return 0;
}