#include "hip/hip_runtime.h"
#include "MatrixOperation.cuh"

#include <hipsparse.h>

template<class T,class Ind> __global__
void CUDA_matrixSum_kernel(int M, int N, int totalANnz, T *dCscValA, Ind *dCscRowPtrA, Ind *dCscColIndA, float* result, bool isCol) {
	LARGEINDEX id = threadIdx.x + blockIdx.x*blockDim.x;
	LARGEINDEX step = gridDim.x*blockDim.x;
	if (isCol) {
		//Col sum
		for (LARGEINDEX colID = id; colID < N; colID = colID + step) {
			int nonzero = dCscColIndA[colID + 1] - dCscColIndA[colID];
			for (LARGEINDEX i = 0; i < nonzero; i++) {
				result[colID] = result[colID] + dCscValA[dCscColIndA[colID] + i];
			}
		}
	}
	else {
		//Row sum
		LARGEINDEX rowID;
		for (LARGEINDEX colID = id; colID < N; colID = colID + step) {
			int nonzero = dCscColIndA[colID + 1] - dCscColIndA[colID];
			for (LARGEINDEX i = 0; i < nonzero; i++) {
				rowID = dCscRowPtrA[dCscColIndA[colID] + i];
				atomicAdd(result + rowID,(float) dCscValA[dCscColIndA[colID] + i]);
			}
		}
	}
}




template<class T>
void CUDA_matrixSum(int M, int N, int totalANnz, T *dCscValA, int *dCscRowPtrA, int *dCscColIndA,T* result, bool isCol) {
	
	hipsparseHandle_t handle=0;
	hipsparseCreate(&handle);
	int K1;
	int K2;
	hipsparseOperation_t op;
	if (isCol) {
		//Col sum
		op = hipsparseOperation_t::HIPSPARSE_OPERATION_NON_TRANSPOSE;
		K1 = M;
		K2 = N;
	}
	else {
		//Row sum
		op = hipsparseOperation_t::HIPSPARSE_OPERATION_TRANSPOSE;
		K1 = N;
		K2 = M;
	}

	T *B, *dB;
	T *dC;
	B = new T[K1];
	for (size_t i = 0; i < K1; ++i) {
		B[i] = 1;
	}

	hipMalloc((void **)&dB, sizeof(*B) * K1);
	hipMalloc((void **)&dC, sizeof(*dC) * K2);

	hipMemcpy(dB, B, sizeof(*B) * K1, hipMemcpyHostToDevice);

	hipsparseMatDescr_t Adescr = 0;
	hipsparseCreateMatDescr(&Adescr);
	hipsparseSetMatType(Adescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(Adescr, HIPSPARSE_INDEX_BASE_ZERO);

	T alpha = 1;
	T beta = 0;

	clock_t begin = clock();
	hipsparseDcsrmv(handle, op, N, M,
		totalANnz, &alpha, Adescr, dCscValA, dCscColIndA, dCscRowPtrA, dB, &beta, dC);

	hipDeviceSynchronize();
	clock_t end = clock();
	double timeSec = (end - begin) / static_cast<double>(CLOCKS_PER_SEC);
	//std::cout << "Elapsed time: " << timeSec << std::endl;
	// Copy the result vector back to the host
	hipMemcpy(result, dC, sizeof(*result) * K2, hipMemcpyDeviceToHost);

	//print_partial_matrix("C:", C, 1, K2, 1, K2);

	delete[](B);
	hipFree(dB);
	hipFree(dC);
//	hipFree(dANnzPerCol);
	hipsparseDestroyMatDescr(Adescr);
	hipsparseDestroy(handle);
}













template __global__ void CUDA_matrixSum_kernel(int M, int N, int totalANnz, float *dCscValA, LARGEINDEX *dCscRowPtrA, LARGEINDEX *dCscColIndA, float* result, bool isCol = false);
template __global__ void CUDA_matrixSum_kernel(int M, int N, int totalANnz, float *dCscValA, int *dCscRowPtrA, int *dCscColIndA, float* result, bool isCol = false);
template __global__ void CUDA_matrixSum_kernel(int M, int N, int totalANnz, double *dCscValA, LARGEINDEX *dCscRowPtrA, LARGEINDEX *dCscColIndA, float* result, bool isCol = false);
template void CUDA_matrixSum(int M, int N, int totalANnz, double *dCscValA, int *dCscRowPtrA, int *dCscColIndA, double* result, bool isCol = false);