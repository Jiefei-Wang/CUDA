#include "SparseMatrix.cuh"
template<class T> __alldev__
SparseMatrix<T>::SparseMatrix(LARGEINDEX * size, LARGEINDEX * offset, Deviceloc location)
{
	 matrixSize = size[0];
	 rowIndexLen = size[1];
	 colIndexLen = size[2];
	 matrixRowDim = size[3];
	 matrixColDim = size[4];
	 this->offset = *offset;
	switch (location) {
	case Deviceloc::Host:
		createHostMatrix(matrixSize, rowIndexLen, colIndexLen);
		break;
	case Deviceloc::Device:
		createDevMatrix(matrixSize, rowIndexLen, colIndexLen);
		break;
	case Deviceloc::Auto:
#ifndef  __CUDA_ARCH__
		createHostMatrix(matrixSize, rowIndexLen, colIndexLen);
#else
		createDevMatrix(matrixSize, rowIndexLen, colIndexLen);
#endif
		break;
	}
}
template <class T> __alldev__
SparseMatrix<T>::SparseMatrix(T* dataFrame, LARGEINDEX* rowInd, LARGEINDEX* colInd, LARGEINDEX* size, LARGEINDEX * offset, Deviceloc location = Deviceloc::Auto) {
	 matrixSize = size[0];
	 rowIndexLen = size[1];
	 colIndexLen = size[2];
	 matrixRowDim = size[3];
	 matrixColDim = size[4];

	 this->offset = *offset;

	switch (location) {
	case Deviceloc::Host:
		matrixData = dataFrame;
		this->rowInd = rowInd;
		this->colInd = colInd;
		break;
	case Deviceloc::Device:
		dev_matrixData = dataFrame;
		dev_rowInd = rowInd;
		dev_colInd = colInd;
		break;
	case Deviceloc::Auto:
#ifndef  __CUDA_ARCH__
		matrixData = dataFrame;
		this->rowInd = rowInd;
		this->colInd = colInd;
#else
		dev_matrixData = dataFrame;
		dev_rowInd = rowInd;
		dev_colInd = colInd;
#endif
		break;
	}
}

template<class T>
SparseMatrix<T>::SparseMatrix(LARGEINDEX * address)
{
	
	matrixSize = address[3];
	rowIndexLen = address[4];
	colIndexLen = address[5];
	matrixRowDim = address[6];
	matrixColDim = address[7];
	this->offset = address[8];
	dev_matrixData = reinterpret_cast<T*>(address[0]);
	dev_rowInd = reinterpret_cast<LARGEINDEX*>(address[1]);
	dev_colInd = reinterpret_cast<LARGEINDEX*>(address[2]);
}

template<class T>
void SparseMatrix<T>::HostToDevice()
{
	if (matrixData != nullptr) {
		if (dev_matrixData == nullptr) {
			delDevMatrix();
			createDevMatrix(matrixSize, rowIndexLen, colIndexLen);
		}
		errorHandle(hipMemcpy(dev_matrixData, matrixData, (matrixSize) * sizeof(T)
			, hipMemcpyHostToDevice), std::string("Error in matrix synchronization to device"));	
		errorHandle(hipMemcpy(dev_rowInd, rowInd, (rowIndexLen) * sizeof(double)
			, hipMemcpyHostToDevice), std::string("Error in matrix synchronization to device"));
		errorHandle(hipMemcpy(dev_colInd, colInd, (colIndexLen) * sizeof(double)
			, hipMemcpyHostToDevice), std::string("Error in matrix synchronization to device"));
	}
	else {
		errorPrint("Error in matrix synchronization: The host matrix does not exist");
	}
}

template<class T>
void SparseMatrix<T>::deviceToHost()
{
	if (dev_matrixData != nullptr) {
		if (matrixData == nullptr) {
			delHostMatrix();
			createHostMatrix(matrixSize, rowIndexLen,colIndexLen);
		}
		errorHandle(hipMemcpy(matrixData,dev_matrixData, (matrixSize ) * sizeof(T), hipMemcpyDeviceToHost), std::string("Error in matrix synchronization to host"));
		errorHandle(hipMemcpy(rowInd, dev_rowInd, (rowIndexLen) * sizeof(double), hipMemcpyDeviceToHost), std::string("Error in matrix synchronization to host"));
		errorHandle(hipMemcpy(colInd, dev_colInd, (colIndexLen) * sizeof(double), hipMemcpyDeviceToHost), std::string("Error in matrix synchronization to host"));
	}
	else {
		errorPrint("Error in matrix synchronization: The device matrix does not exist");
	}
}



template<class T>
void SparseMatrix<T>::delHostMatrix()
{
	if (matrixData != nullptr) {
		delete[] matrixData;
		matrixData = nullptr;
		delete[] rowInd;
		rowInd = nullptr;
		delete[] colInd;
		colInd = nullptr;
	}
}

template<class T>
void SparseMatrix<T>::delDevMatrix()
{
	if (dev_matrixData != nullptr) {
		errorHandle(hipFree(dev_matrixData), std::string("Error in deleting device matrix"));
		errorHandle(hipFree(dev_rowInd), std::string("Error in deleting device matrix"));
		errorHandle(hipFree(dev_colInd), std::string("Error in deleting device matrix"));
		dev_matrixData = nullptr;
	}
}

template<class T>
void SparseMatrix<T>::createHostMatrix(LARGEINDEX matrixSize, LARGEINDEX rowIndexLen, LARGEINDEX colIndexLen)
{
	this->matrixSize = matrixSize;
	this->rowIndexLen = rowIndexLen;
	this->colIndexLen = colIndexLen;
	matrixData = new T[matrixSize];
	rowInd = new LARGEINDEX[rowIndexLen];
	colInd = new LARGEINDEX[colIndexLen];
}
template<class T>
void SparseMatrix<T>::createDevMatrix(LARGEINDEX matrixSize, LARGEINDEX rowIndexLen, LARGEINDEX colIndexLen)
{
	this->matrixSize = matrixSize;
	this->rowIndexLen = rowIndexLen;
	this->colIndexLen = colIndexLen;
	errorHandle(hipMalloc(&dev_matrixData,
		(matrixSize) *sizeof(T)), std::string("Error in create device matrix"));
	errorHandle(hipMalloc(&dev_rowInd,
		(rowIndexLen) * sizeof(LARGEINDEX)), std::string("Error in create device matrix"));
	errorHandle(hipMalloc(&dev_colInd,
		(colIndexLen) * sizeof(LARGEINDEX)), std::string("Error in create device matrix"));
}

template<class T>
void SparseMatrix<T>::setPackedInfo(LARGEINDEX* Info)
{
	Info[0] =(LARGEINDEX) dev_matrixData;
	Info[1] = (LARGEINDEX)dev_rowInd;
	Info[2] = (LARGEINDEX)dev_colInd;
	Info[3] = matrixSize;
	Info[4] = rowIndexLen;
	Info[5] = colIndexLen;
	Info[6] = matrixRowDim;
	Info[7] = matrixColDim;
	Info[8] = offset;
}
using namespace std;
template<class T>
void SparseMatrix<T>::print()
{
	cout << "Matrix Data: ";
	for (LARGEINDEX i = 0; i < matrixSize; i++) {
		cout << matrixData[i] << " ";
	}
	cout << endl;
	cout << "Row index: ";
	for (LARGEINDEX i = 0; i < rowIndexLen; i++) {
		cout << rowInd[i] << " ";
	}
	cout << endl;
	cout << "Column index: ";
	for (LARGEINDEX i = 0; i < colIndexLen; i++) {
		cout << colInd[i] << " ";
	}
	cout << endl;
	cout << "Matrix size: " << matrixSize <<
		", Row size: " << rowIndexLen <<
		", Column size: " << colIndexLen <<
		", Offset: " << offset << endl;
	cout <<"Row dimension: " << matrixRowDim <<
		", Column dimension: " << matrixColDim << endl;
}



