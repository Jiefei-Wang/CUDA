#include "Tools.cuh"

void errorHandle(hipError_t error, std::string msg) {
	static int count = 0;
	if (error != hipSuccess) {
		count++;
		std::cout << count << "." << hipGetErrorString(error);
		if (msg.length() != 0) {
			std::cout << ":"<<msg.c_str()<<std::endl;
		}
	}
}


__host__ __device__
void errorPrint(char * msg)
{
#ifndef  __CUDA_ARCH__
	std::cout << msg << std::endl;
#endif
}


LARGEINDEX *  getIndexFromR(double* source, LARGEINDEX length) {
	LARGEINDEX * target = new LARGEINDEX[length];
	for (LARGEINDEX i = 0; i < length; i++) {
		target[i] = source[i];
	}
	return(target);
}

template<class T>
void fillWithNum(T* target, T number, int n) {
	for (int i = 0; i < n; i++) {
		target[i] = number;
	}
}

template<class T>
void print_partial_matrix(char* title, T *M, int nrows, int ncols, int max_row,
	int max_col)
{
	std::cout << title << std::endl;
	int row, col;
	if (max_row == -1) max_row = nrows;
	if (max_col == -1) max_col = ncols;
	for (row = 0; row < max_row; row++)
	{
		for (col = 0; col < max_col; col++)
		{
			std::cout << M[row + col*nrows] << " ";
		}
		printf("...\n");
	}
	printf("...\n");
}


template void transformData(double* target, LARGEINDEX* source, LARGEINDEX length);
template void transformData(LARGEINDEX* target, double* source, LARGEINDEX length);
template void transformData(LARGEINDEX* target, LARGEINDEX* source, LARGEINDEX length);

template void fillWithNum(double* target, double number, int n);
template void fillWithNum(float* target, float number, int n);
template void fillWithNum(int* target, int number, int n);

template void print_partial_matrix(char* title, double *M, int nrows, int ncols, int max_row,
	int max_col) ;
template void print_partial_matrix(char* title, float *M, int nrows, int ncols, int max_row,
	int max_col);
template void print_partial_matrix(char* title, int *M, int nrows, int ncols, int max_row,
	int max_col);


