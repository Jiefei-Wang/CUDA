#include "Rfuncs.cuh"
using namespace std;
template <class T> void print(T* a,int length, char* note) {
	cout << note<<" : ";
	for (int i = 0; i < length-1; i++) {
		cout << a[i] << "," ;
	}
	cout << a[length - 1]<<endl;
}
extern "C" LibExport
void upload(double* dataFrame, double * rowInd, double * colInd, double * size, double* offset, double* address) {
	LARGEINDEX * size_l = getIndexFromR(size, 5);
	LARGEINDEX * rowInd_l = getIndexFromR(rowInd, size_l[1]);
	LARGEINDEX * colInd_l = getIndexFromR(colInd, size_l[2]);
	LARGEINDEX * offset_l = getIndexFromR(offset, 1);

	SparseMatrix<double> dataMatrix(dataFrame, rowInd_l, colInd_l, size_l, offset_l, Deviceloc::Host);
	dataMatrix.HostToDevice();
	
	dataMatrix.setPackedInfo((LARGEINDEX *)address);
}
extern "C" LibExport
void download(double* data, double * rowInd, double * colInd, double * address) {
	SparseMatrix<double> dataMatrix((LARGEINDEX *)address);
	dataMatrix.deviceToHost();
	transformData(data, dataMatrix.matrixData, dataMatrix.matrixSize);
	transformData(rowInd, dataMatrix.rowInd, dataMatrix.rowIndexLen);
	transformData(colInd, dataMatrix.colInd, dataMatrix.colIndexLen);

	//dataMatrix.print();
}


int * cpyfunc(LARGEINDEX* src, int n) {
	int* tmp = new int[n];
	LARGEINDEX* tmp1 = new LARGEINDEX[n];

	hipMemcpy(tmp1, src, sizeof(*tmp1) * n, hipMemcpyDeviceToHost);
	for (int i = 0; i < n; i++) {
		tmp[i] = tmp1[i];
	}
	int* target = 0;
	hipMalloc((void **)&target, sizeof(*target) * n);
	hipMemcpy(target, tmp, sizeof(*tmp) * n, hipMemcpyHostToDevice);
	delete[] tmp;
	delete[] tmp1;
	return(target);

}
extern "C" LibExport
void colSums( int *direction,double * address, double * result)
{
	int nresult;
	SparseMatrix<double> dataMatrix((LARGEINDEX *)address);
	if (*direction == 1)
		nresult = dataMatrix.matrixRowDim;
	else
		nresult = dataMatrix.matrixColDim;

	fillWithNum(result, 0.0, nresult);

	
	

	float* dev_result = 0;
	float* host_result = new float[nresult];
	fillWithNum(host_result, (float)0, nresult);
	hipMalloc((void **)&dev_result, sizeof(*dev_result) * nresult);
	hipMemcpy(dev_result, host_result, sizeof(*host_result) * nresult, hipMemcpyHostToDevice);

	CUDA_matrixSum_kernel << <BlockNum, ThreadNum >> >(dataMatrix.matrixRowDim, dataMatrix.matrixColDim, dataMatrix.matrixSize,
		dataMatrix.dev_matrixData, dataMatrix.dev_rowInd, dataMatrix.dev_colInd, dev_result, *direction == 2);

	hipDeviceSynchronize();
	hipMemcpy(host_result, dev_result, sizeof(*host_result) * nresult, hipMemcpyDeviceToHost);
	for (int i = 0; i < nresult; i++) {
		result[i] = host_result[i];
	}
	delete[] host_result;
	hipFree(dev_result);
/*
	int* dCscRowPtrA = cpyfunc(dataMatrix.dev_rowInd, dataMatrix.matrixSize);
	int* dCscColIndA = cpyfunc(dataMatrix.dev_colInd, dataMatrix.matrixColDim + 1);

	CUDA_matrixSum(dataMatrix.matrixRowDim, dataMatrix.matrixColDim, dataMatrix.matrixSize,
		dataMatrix.dev_matrixData, dCscRowPtrA, dCscColIndA, result, *direction == 2);
		*/
	//print(result, nresult, "result");


}

